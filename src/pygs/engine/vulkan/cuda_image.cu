#include "pygs/engine/vulkan/cuda_image.h"

#include <hip/hip_runtime.h>

#ifdef _WIN32
#include <windows.h>

#include <vulkan/vulkan_win32.h>
#endif

#include "pygs/engine/vulkan/context.h"

namespace pygs {
namespace vk {
namespace {

hipExternalMemory_t ImportVulkanMemoryObjectFromFileDescriptor(
    int fd, unsigned long long size, bool isDedicated) {
  hipExternalMemory_t extMem = NULL;
  hipExternalMemoryHandleDesc desc = {};
  memset(&desc, 0, sizeof(desc));

  desc.type = hipExternalMemoryHandleTypeOpaqueFd;
  desc.handle.fd = fd;
  desc.size = size;
  if (isDedicated) {
    desc.flags |= hipExternalMemoryDedicated;
  }

  hipImportExternalMemory(&extMem, &desc);

  // Input parameter 'fd' should not be used beyond this point as CUDA has
  // assumed ownership of it
  return extMem;
}

#ifdef _WIN32
hipExternalMemory_t ImportVulkanMemoryObjectFromNTHandle(
    HANDLE handle, unsigned long long size, bool isDedicated) {
  hipExternalMemory_t extMem = NULL;
  hipExternalMemoryHandleDesc desc = {};
  memset(&desc, 0, sizeof(desc));

  desc.type = hipExternalMemoryHandleTypeOpaqueWin32;
  desc.handle.win32.handle = handle;
  desc.size = size;
  if (isDedicated) {
    desc.flags |= hipExternalMemoryDedicated;
  }

  hipImportExternalMemory(&extMem, &desc);

  // Input parameter 'handle' should be closed if it's not needed anymore
  CloseHandle(handle);

  return extMem;
}
#endif

void* MapBufferOntoExternalMemory(hipExternalMemory_t extMem,
                                  unsigned long long offset,
                                  unsigned long long size) {
  void* ptr = NULL;

  hipExternalMemoryBufferDesc desc = {};
  memset(&desc, 0, sizeof(desc));
  desc.offset = offset;
  desc.size = size;
  hipExternalMemoryGetMappedBuffer(&ptr, extMem, &desc);

  // Note: ‘ptr’ must eventually be freed using hipFree()
  return ptr;
}

}  // namespace

class CudaImage::Impl {
 public:
  Impl() = delete;

  Impl(Context context, uint32_t width, uint32_t height)
      : context_(context), width_(width), height_(height) {
#ifdef _WIN32
    constexpr VkExternalMemoryHandleTypeFlagBits handle_type =
        VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_BIT;
#else
    constexpr VkExternalMemoryHandleTypeFlagBits handle_type =
        VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_FD_BIT;
#endif
    VkExternalMemoryImageCreateInfo external_image_info = {
        VK_STRUCTURE_TYPE_EXTERNAL_MEMORY_IMAGE_CREATE_INFO};
    external_image_info.handleTypes = handle_type;

    VkImageCreateInfo image_info = {VK_STRUCTURE_TYPE_IMAGE_CREATE_INFO};
    image_info.pNext = &external_image_info;
    image_info.imageType = VK_IMAGE_TYPE_2D;
    image_info.format = VK_FORMAT_R32G32B32A32_SFLOAT;
    image_info.extent = {width, height, 1};
    image_info.mipLevels = 1;
    image_info.arrayLayers = 1;
    image_info.samples = VK_SAMPLE_COUNT_1_BIT;
    image_info.tiling = VK_IMAGE_TILING_LINEAR;
    image_info.usage =
        VK_IMAGE_USAGE_TRANSFER_SRC_BIT | VK_IMAGE_USAGE_TRANSFER_DST_BIT;
    image_info.initialLayout = VK_IMAGE_LAYOUT_UNDEFINED;
    vkCreateImage(context.device(), &image_info, NULL, &image_);

    // Memory
    // TODO: allocate large memory
    VkExportMemoryAllocateInfo external_memory_info = {
        VK_STRUCTURE_TYPE_EXPORT_MEMORY_ALLOCATE_INFO};
    external_memory_info.handleTypes = handle_type;

    const VkDeviceSize size =
        static_cast<VkDeviceSize>(width) * height * 4 * sizeof(float);
    VkMemoryAllocateInfo memory_info = {VK_STRUCTURE_TYPE_MEMORY_ALLOCATE_INFO};
    memory_info.pNext = &external_memory_info;
    memory_info.memoryTypeIndex = 0;  // TODO
    memory_info.allocationSize = size;
    vkAllocateMemory(context_.device(), &memory_info, NULL, &memory_);

    vkBindImageMemory(context.device(), image_, memory_, 0);

#ifdef _WIN32
    VkMemoryGetWin32HandleInfoKHR handle_info = {
        VK_STRUCTURE_TYPE_MEMORY_GET_WIN32_HANDLE_INFO_KHR};
    handle_info.memory = memory_;
    handle_info.handleType = handle_type;
    HANDLE handle;
    context_.GetMemoryWin32HandleKHR(&handle_info, &handle);

    hipExternalMemory_t ext_mem =
        ImportVulkanMemoryObjectFromNTHandle(handle, size, false);
#else
    VkMemoryGetFdInfoKHR fd_info = {VK_STRUCTURE_TYPE_MEMORY_GET_FD_INFO_KHR};
    fd_info.memory = memory_;
    fd_info.handleType = handle_type;
    int fd = -1;
    context_.GetMemoryFdKHR(&fd_info, &fd);

    hipExternalMemory_t ext_mem =
        ImportVulkanMemoryObjectFromFileDescriptor(fd, size, false);
#endif

    map_ = MapBufferOntoExternalMemory(ext_mem, 0, size);
  }

  ~Impl() {
    vkDestroyImage(context_.device(), image_, NULL);
    vkFreeMemory(context_.device(), memory_, NULL);
    hipFree(map_);
  }

  VkImage image() const noexcept { return image_; }
  uint32_t width() const noexcept { return width_; }
  uint32_t height() const noexcept { return height_; }
  void* map() noexcept { return map_; }
  const void* map() const noexcept { return map_; }

 private:
  Context context_;
  VkImage image_ = VK_NULL_HANDLE;
  VkDeviceMemory memory_ = VK_NULL_HANDLE;

  uint32_t width_ = 0;
  uint32_t height_ = 0;
  void* map_ = nullptr;
};

CudaImage::CudaImage() = default;

CudaImage::CudaImage(Context context, uint32_t width, uint32_t height)
    : impl_(std::make_shared<Impl>(context, width, height)) {}

CudaImage::~CudaImage() = default;

VkImage CudaImage::image() const { return impl_->image(); }

uint32_t CudaImage::width() const { return impl_->width(); }

uint32_t CudaImage::height() const { return impl_->height(); }

void* CudaImage::map() { return impl_->map(); }

const void* CudaImage::map() const { return impl_->map(); }

}  // namespace vk
}  // namespace pygs
