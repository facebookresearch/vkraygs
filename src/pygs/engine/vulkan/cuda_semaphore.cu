#include "cuda_semaphore.h"

namespace pygs {
namespace vk {
namespace {

hipExternalSemaphore_t ImportVulkanSemaphoreObjectFromFileDescriptor(int fd) {
  hipExternalSemaphore_t extSem = NULL;
  hipExternalSemaphoreHandleDesc desc = {};
  memset(&desc, 0, sizeof(desc));

  desc.type = hipExternalSemaphoreHandleTypeOpaqueFd;
  desc.handle.fd = fd;
  hipImportExternalSemaphore(&extSem, &desc);

  // Input parameter 'fd' should not be used beyond this point as CUDA has
  // assumed ownership of it
  return extSem;
}

void SignalExternalSemaphore(hipExternalSemaphore_t extSem,
                             hipStream_t stream) {
  hipExternalSemaphoreSignalParams params = {};
  memset(&params, 0, sizeof(params));
  hipSignalExternalSemaphoresAsync(&extSem, &params, 1, stream);
}

}  // namespace

class CudaSemaphore::Impl {
 public:
  Impl() = delete;

  Impl(Context context) : context_(context) {
    VkExportSemaphoreCreateInfo external_semaphore_info = {
        VK_STRUCTURE_TYPE_EXPORT_SEMAPHORE_CREATE_INFO};
    external_semaphore_info.handleTypes =
        VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_FD_BIT;

    VkSemaphoreCreateInfo semaphore_info = {
        VK_STRUCTURE_TYPE_SEMAPHORE_CREATE_INFO};
    semaphore_info.pNext = &external_semaphore_info;
    vkCreateSemaphore(context.device(), &semaphore_info, NULL, &semaphore_);

    VkSemaphoreGetFdInfoKHR fd_info = {
        VK_STRUCTURE_TYPE_SEMAPHORE_GET_FD_INFO_KHR};
    fd_info.semaphore = semaphore_;
    fd_info.handleType = VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_FD_BIT;
    int fd = -1;
    context_.GetSemaphoreFdKHR(&fd_info, &fd);

    cuda_semaphore_ = ImportVulkanSemaphoreObjectFromFileDescriptor(fd);
  }

  ~Impl() { vkDestroySemaphore(context_.device(), semaphore_, NULL); }

  VkSemaphore semaphore() const noexcept { return semaphore_; }

  void signal(hipStream_t stream) {
    SignalExternalSemaphore(cuda_semaphore_, stream);
  }

 private:
  Context context_;
  VkSemaphore semaphore_ = VK_NULL_HANDLE;
  hipExternalSemaphore_t cuda_semaphore_ = nullptr;
};

CudaSemaphore::CudaSemaphore() = default;

CudaSemaphore::CudaSemaphore(Context context)
    : impl_(std::make_shared<Impl>(context)) {}

CudaSemaphore::~CudaSemaphore() = default;

VkSemaphore CudaSemaphore::semaphore() const { return impl_->semaphore(); }

void CudaSemaphore::signal(hipStream_t stream) { impl_->signal(stream); }

}  // namespace vk
}  // namespace pygs
